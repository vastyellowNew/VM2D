#include "hip/hip_runtime.h"
/*--------------------------------*- VM2D -*-----------------*---------------*\
| ##  ## ##   ##  ####  #####   |                            | Version 1.6    |
| ##  ## ### ### ##  ## ##  ##  |  VM2D: Vortex Method       | 2019/10/28     |
| ##  ## ## # ##    ##  ##  ##  |  for 2D Flow Simulation    *----------------*
|  ####  ##   ##   ##   ##  ##  |  Open Source Code                           |
|   ##   ##   ## ###### #####   |  https://www.github.com/vortexmethods/VM2D  |
|                                                                             |
| Copyright (C) 2017-2019 Ilia Marchevsky, Kseniia Kuzmina, Evgeniya Ryatina  |
*-----------------------------------------------------------------------------*
| File name: cuLib2D.cu                                                       |
| Info: Source code of VM2D                                                   |
|                                                                             |
| This file is part of VM2D.                                                  |
| VM2D is free software: you can redistribute it and/or modify it             |
| under the terms of the GNU General Public License as published by           |
| the Free Software Foundation, either version 3 of the License, or           |
| (at your option) any later version.                                         |
|                                                                             |
| VM is distributed in the hope that it will be useful, but WITHOUT           |
| ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or       |
| FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License       |
| for more details.                                                           |
|                                                                             |
| You should have received a copy of the GNU General Public License           |
| along with VM2D.  If not, see <http://www.gnu.org/licenses/>.               |
\*---------------------------------------------------------------------------*/


/*!
\file
\brief Файл с реализацией функций библиотеки cuLib для работы с CUDA
\author Марчевский Илья Константинович
\author Кузьмина Ксения Сергеевна
\author Рятина Евгения Павловна
\version 1.6   
\date 28 октября 2019 г.
*/

#include <iostream>

#include "cuLib2D.cuh"

#include "hip/hip_runtime.h"
#include "Gpudefs.h"


__device__ __constant__ size_t sizeVort;
__device__ __constant__ size_t posR;
__device__ __constant__ size_t posG;

__device__ __constant__ double accelCoeff;

__device__ __constant__ double maxGamma;
__device__ __constant__ double collapseRightBorder;
__device__ __constant__ double collapseScale;


#define invdpi (0.15915494309189533576888376337251)
#define pi (3.1415926535897932384626433832795)


/// \brief Способ сглаживания скорости вихря (вихрь Рэнкина или вихрь Ламба)
__device__ double CUboundDenom(double r2, double eps2)
{
#ifndef LAMBVORTEX
	return fmax(r2, eps2);
#else
	if (r2 > eps2)
		return fmax(r2, eps2);
	else
		return (r2 < 1e-10) ? 1e-10 : r2  / (1.0 - exp(-6.0*r2 / eps2));
#endif
}



//Ниже - ядра (__global__), затем - "обычные" функции (__host__)

__global__ void CU_WakeToZero(size_t nvt, double* vt)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	vt[i*sizeVort + posR + 0] = 0.0;
	vt[i*sizeVort + posR + 1] = 0.0;
	vt[i*sizeVort + posG]     = 0.0;
}


__global__ void CU_calc_conv_epsast(
	size_t disp, size_t len, double* pt,
	size_t nvt, double* vt,
	size_t nsr, double* sr,
	double eps2, double minRd,
	double* vel, double* rad,
	size_t nAfls, size_t* nVtxs, double** ptrVtxs)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];

	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	double velx = 0.0;
	double vely = 0.0;

	double dx, dy, dr2;
	double izn;

	double d_1 = 1e+5;
	double d_2 = 1e+5;
	double d_3 = 1e+5;
	double d_0 = 1e+5;
	double dst23, dst12, dst01;

	//vortices
	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];

		__syncthreads();
	
		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			dx = ptx - shx[q];
			dy = pty - shy[q];
			dr2 = dx*dx + dy*dy;

			izn = shg[q] / CUboundDenom(dr2, eps2); //Сглаживать надо!!!

			velx -= dy * izn;
			vely += dx * izn;

			if (d_3 > dr2) 
			{
				dst23 = fmin(dr2, d_2);
				d_3 = fmax(dr2, d_2);
				
				dst12 = fmin(dst23, d_1);
				d_2 = fmax(dst23, d_1);
				
				dst01 = fmin(dst12, d_0);
				d_1 = fmax(dst12, d_0);
				d_0 = dst01;
			}
		}
		__syncthreads();
	}

	//sources
	for (size_t j = 0; j < nsr; j += CUBLOCK)
	{
		shx[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posG + 0] * accelCoeff;

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nsr)
			{
				dx = ptx - shx[q];
				dy = pty - shy[q];
				dr2 = dx * dx + dy * dy;

				izn = shg[q] / CUboundDenom(dr2, eps2); //Сглаживать надо!!!

				velx += dx * izn;
				vely += dy * izn;
			}
		}
		__syncthreads();
	}

	
	for (size_t q = 0; q < nAfls; ++q)
	for (size_t j = 0; j < nVtxs[q]; j += CUBLOCK)
	{
		shx[threadIdx.x] = ptrVtxs[q][(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = ptrVtxs[q][(j + threadIdx.x)*sizeVort + posR + 1];
				
		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			dx = ptx - shx[q];
			dy = pty - shy[q];
			dr2 = dx*dx + dy*dy;

			if (d_3 > dr2) 
			{
				dst23 = fmin(dr2, d_2);
				d_3 = fmax(dr2, d_2);

				dst12 = fmin(dst23, d_1);
				d_2 = fmax(dst23, d_1);

				dst01 = fmin(dst12, d_0);
				d_1 = fmax(dst12, d_0);
				d_0 = dst01;
			}
		}
		__syncthreads();
	}

	vel[2 * locI + 0] = velx * invdpi;
	vel[2 * locI + 1] = vely * invdpi;
	rad[locI] = fmax( sqrt( (d_1 + d_2 + d_3) / 3.0), minRd);
}



__global__ void CU_calc_only_epsast(
	size_t disp, size_t len, double* pt,
	size_t nvt, double* vt,
	size_t nsr, double* sr,
	double eps2, double minRd,
	double* rad,
	size_t nAfls, size_t* nVtxs, double** ptrVtxs)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];

	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	double dx, dy, dr2;

	double d_1 = 1e+5;
	double d_2 = 1e+5;
	double d_3 = 1e+5;
	double d_0 = 1e+5;
	double dst23, dst12, dst01;

	//vortices
	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			dx = ptx - shx[q];
			dy = pty - shy[q];
			dr2 = dx * dx + dy * dy;
			
			if (d_3 > dr2)
			{
				dst23 = fmin(dr2, d_2);
				d_3 = fmax(dr2, d_2);

				dst12 = fmin(dst23, d_1);
				d_2 = fmax(dst23, d_1);

				dst01 = fmin(dst12, d_0);
				d_1 = fmax(dst12, d_0);
				d_0 = dst01;
			}
		}
		__syncthreads();
	}
	
	for (size_t q = 0; q < nAfls; ++q)
		for (size_t j = 0; j < nVtxs[q]; j += CUBLOCK)
		{
			shx[threadIdx.x] = ptrVtxs[q][(j + threadIdx.x)*sizeVort + posR + 0];
			shy[threadIdx.x] = ptrVtxs[q][(j + threadIdx.x)*sizeVort + posR + 1];

			__syncthreads();

			for (size_t q = 0; q < CUBLOCK; ++q)
			{
				dx = ptx - shx[q];
				dy = pty - shy[q];
				dr2 = dx * dx + dy * dy;

				if (d_3 > dr2)
				{
					dst23 = fmin(dr2, d_2);
					d_3 = fmax(dr2, d_2);

					dst12 = fmin(dst23, d_1);
					d_2 = fmax(dst23, d_1);

					dst01 = fmin(dst12, d_0);
					d_1 = fmax(dst12, d_0);
					d_0 = dst01;
				}
			}
			__syncthreads();
		}

	rad[locI] = fmax(sqrt((d_1 + d_2 + d_3) / 3.0), minRd);
}




__global__ void CU_calc_conv_From_Panels(
	size_t disp, size_t len, double* pt,
	size_t npnl, double* r, double* freegamma, double* attgamma, double* attsource,
	double eps2, 
	double* vel)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];

	__shared__ double shdx[CUBLOCK];
	__shared__ double shdy[CUBLOCK];

	__shared__ double shlen[CUBLOCK];


	__shared__ double shfreegamma[CUBLOCK];
	__shared__ double shattgamma[CUBLOCK];
	__shared__ double shattsource[CUBLOCK];

	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	double velx = 0.0;
	double vely = 0.0;

	double sx, sy, px, py, s2, p2, alpha, lambda, taux, tauy, psix, psiy;
	int rindexnext;

	for (size_t j = 0; j < npnl; j += CUBLOCK)
	{
		shx[threadIdx.x] = r[(j + threadIdx.x)*2 + 0];
		shy[threadIdx.x] = r[(j + threadIdx.x)*2 + 1];
		
		rindexnext = ((j + threadIdx.x + 1) < npnl) ? j + threadIdx.x + 1 : 0;

		shdx[threadIdx.x] = r[rindexnext * 2 + 0] - shx[threadIdx.x];
		shdy[threadIdx.x] = r[rindexnext * 2 + 1] - shy[threadIdx.x];

		shlen[threadIdx.x] = sqrt(shdx[threadIdx.x] * shdx[threadIdx.x] + shdy[threadIdx.x] * shdy[threadIdx.x]);
		
		shfreegamma[threadIdx.x] = freegamma[j + threadIdx.x];
		shattgamma[threadIdx.x] = attgamma[j + threadIdx.x];
		shattsource[threadIdx.x] = attsource[j + threadIdx.x];

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < npnl)
			{
				sx = ptx - shx[q];
				sy = pty - shy[q];

				px = sx - shdx[q];
				py = sy - shdy[q];

				alpha = atan2(px*sy - py*sx, px*sx + py*sy);

				s2 = sx*sx + sy*sy;
				p2 = px*px + py*py;
				
				if ((s2 > 1e-16) && (p2 > 1e-16))
					lambda = 0.5*log(s2 / p2);
				else 
					lambda = 0.0;

				taux = shdx[q] / shlen[q];
				tauy = shdy[q] / shlen[q];

				psix = alpha*tauy + lambda*taux;
				psiy = -alpha*taux + lambda*tauy;

				//kpsix = -psiy;
				//kpsiy =  psix;

				velx += (shfreegamma[q] + shattgamma[q]) * (-psiy) + shattsource[q] * psix;
				vely += (shfreegamma[q] + shattgamma[q]) * (psix)+shattsource[q] * psiy;
			}
		}
		__syncthreads();
	}	

	vel[2 * locI + 0] = velx * invdpi;
	vel[2 * locI + 1] = vely * invdpi;
}


__global__ void CU_calc_I1I2(
	size_t disp, size_t len, double* pt,
	size_t nvt, double* vt,
	double* i1, double* i2,
	double* rd)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];
	
	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = rd[i];

	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double dx, dy, dr;
	double expr, exprdivdr;
	
	double diffRadius = 8.0*rdi;

	double left = ptx - diffRadius;
	double right = ptx + diffRadius;

	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];
		
		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if ((shx[q] < right) && (shx[q] > left))
			{
				dx = ptx - shx[q];
				dy = pty - shy[q];

				dr = sqrt(dx*dx + dy*dy);

				if ((dr < diffRadius) && (dr > 1e-10))
				{
					expr = shg[q] * exp(-dr / rdi);
					exprdivdr = expr / dr;
					val1 += expr;
					val2x += exprdivdr * dx;
					val2y += exprdivdr * dy;
				}//if (rij>1e-10)
			}
		}
		__syncthreads();
	}
	i1[locI] = val1;
	i2[2 * locI + 0] = val2x;
	i2[2 * locI + 1] = val2y;
}



__global__ void CU_calc_I1I2mesh(
	size_t disp, size_t len, double* pt,
	size_t nvt, double* vt,
	double* i1, double* i2,
	double* rd, int* dev_ptr_mesh)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];
	__shared__ int shmshx[CUBLOCK];
	__shared__ int shmshy[CUBLOCK];

	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = rd[i];

	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double dx, dy, dr;
	double expr, exprdivdr;
	
	double diffRadius = 8.0*rdi;

	int imshx = dev_ptr_mesh[2 * i + 0];
	int imshy = dev_ptr_mesh[2 * i + 1];

	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];

		shmshx[threadIdx.x] = dev_ptr_mesh[(j + threadIdx.x)*2 + 0];
		shmshy[threadIdx.x] = dev_ptr_mesh[(j + threadIdx.x)*2 + 1];

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if ((abs(imshx-shmshx[q])<25) && (abs(imshy - shmshy[q])<25))
			{
				dx = ptx - shx[q];
				dy = pty - shy[q];

				dr = sqrt(dx*dx + dy*dy);

				if ((dr < diffRadius) && (dr > 1e-10))
				{
					expr = shg[q] * exp(-dr / rdi);
					exprdivdr = expr / dr;
					val1 += expr;
					val2x += exprdivdr * dx;
					val2y += exprdivdr * dy;
				}//if (rij>1e-10)
			}
		}
		__syncthreads();
	}
	i1[locI] = val1;
	i2[2 * locI + 0] = val2x;
	i2[2 * locI + 1] = val2y;
}



__global__ void CU_calc_I1I2FromPanels(
	size_t disp, size_t len, double* pt,
	size_t npnl, double* r, double* freegamma,
	double* i1, double* i2,
	double* rd)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shxp1[CUBLOCK];
	__shared__ double shyp1[CUBLOCK];

	__shared__ double shtaux[CUBLOCK];
	__shared__ double shtauy[CUBLOCK];

	__shared__ double shlen[CUBLOCK];
	__shared__ double shptG[CUBLOCK];

	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = rd[i];

	double val1 = 0.0;
	double val2x = 0.0;
	double val2y = 0.0;

	double x0, y0, mn;

	double dx, dy, dr;
	double expr, exprdivdr;

	double diffRadius = 8.0*rdi;

	double left = ptx - diffRadius;
	double right = ptx + diffRadius;

	const int nQuadPt = 3;

	int rindexnext;

	for (size_t j = 0; j < npnl; j += CUBLOCK)
	{
		shx[threadIdx.x] = r[(j + threadIdx.x)*2 + 0];
		shy[threadIdx.x] = r[(j + threadIdx.x)*2 + 1];		
	
		rindexnext = ((j + threadIdx.x + 1) < npnl) ? j + threadIdx.x + 1 : 0;
		
		shxp1[threadIdx.x] = r[rindexnext * 2 + 0];
		shyp1[threadIdx.x] = r[rindexnext * 2 + 1];

		shtaux[threadIdx.x] = shxp1[threadIdx.x] - shx[threadIdx.x];
		shtauy[threadIdx.x] = shyp1[threadIdx.x] - shy[threadIdx.x];

		shlen[threadIdx.x] = sqrt(shtaux[threadIdx.x] * shtaux[threadIdx.x] + shtauy[threadIdx.x] * shtauy[threadIdx.x] );

		shptG[threadIdx.x] = freegamma[j + threadIdx.x] * shlen[threadIdx.x] / nQuadPt;

		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			double xcnt = 0.5*(shx[q] + shxp1[q]);
			if ((xcnt < right) && (xcnt > left))
			{
				for (int s = 0; s < nQuadPt; ++s)
				{
					mn = (s + 0.5) / nQuadPt;
					x0 = shx[q] + shtaux[q] * mn;
					y0 = shy[q] + shtauy[q] * mn;


					dx = ptx - x0;
					dy = pty - y0;

					dr = sqrt(dx*dx + dy*dy);

					if ((dr < diffRadius) && (dr > 1e-10))
					{
						expr = shptG[q] * exp(-dr / rdi);
						exprdivdr = expr / dr;
						val1 += expr;
						val2x += exprdivdr * dx;
						val2y += exprdivdr * dy;
					}//if (rij>1e-10)
				}
			}
		}
		__syncthreads();
	}
	i1[locI] = val1;
	i2[2 * locI + 0] = val2x;
	i2[2 * locI + 1] = val2y;
}




__global__ void CU_calc_I0I3(
	size_t disp, size_t len, double* pt,
	size_t nvt, double* vt,
	double* i0, double* i3,
	double* rd)
{
	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];
	double rdi = rd[i];

	double val0 = 0.0;
	double val3x = 0.0;
	double val3y = 0.0;

	double iDDomRad = 1.0 / rdi;

	double qx, qy, d;
	double begx, begy, endx, endy;
	double lenj, lenj_m;
	double taux, tauy;
	double s;
	double normx, normy;
	double v0x, v0y;
	double hx, hy;
	double xix, xiy, lxi;
	double expon;
	double mnx, mny;
	int new_n;
	double xi_mx, xi_my, lxi_m;
	double mnog1;

	int vtindexnext;

	for (size_t j = 0; j < nvt; ++j)
	{
		begx = vt[j * 2 + 0];
		begy = vt[j * 2 + 1];

		vtindexnext = ((j + 1) < nvt) ? j + 1 : 0;

		endx = vt[vtindexnext * 2 + 0];
		endy = vt[vtindexnext * 2 + 1];

		qx = ptx - 0.5 * (begx + endx);
		qy = pty - 0.5 * (begy + endy);

		lenj = sqrt((endx - begx)*(endx - begx) + (endy - begy)*(endy - begy));

		taux = (endx - begx) / lenj;
		tauy = (endy - begy) / lenj;

		s = qx * taux + qy * tauy;

		normx = tauy;
		normy = -taux;

		d = sqrt(qx*qx + qy*qy);

		if (d < 50.0 * lenj)	//Почему зависит от длины панели???
		{
			v0x = taux * lenj;
			v0y = tauy * lenj;

			if (d > 5.0 * lenj)
			{
				xix = qx * iDDomRad;
				xiy = qy * iDDomRad;
				lxi = sqrt(xix*xix + xiy*xiy);

				expon = exp(-lxi)*lenj;
				mnx = normx*expon;
				mny = normy*expon;

				if (val0 != -pi * rdi)
				{
					val0 += (xix * mnx + xiy * mny) * (lxi + 1.0) / (lxi*lxi);
					val3x += mnx;
					val3y += mny;
				}

				//viscousStress[j] += locPoints[i].g() * expon * iDPIepscol2;
			}
			else if ((d <= 5.0 * lenj) && (d >= 0.1 * lenj))
			{
				//vs = 0.0;
				//new_n = 100;
				new_n = (int)(ceil(5.0 * lenj / d));

				hx = v0x / new_n;
				hy = v0y / new_n;

				for (int m = 0; m < new_n; m++)
				{
					xi_mx = (ptx - (begx + hx * (m + 0.5))) * iDDomRad;
					xi_my = (pty - (begy + hy * (m + 0.5))) * iDDomRad;

					lxi_m = sqrt(xi_mx*xi_mx + xi_my*xi_my);

					lenj_m = lenj / new_n;
					expon = exp(-lxi_m)*lenj_m;
					

					mnx = normx * expon;
					mny = normy * expon;

					if (val0 != -pi * rdi)
					{
						val0 += (xi_mx*mnx + xi_my*mny) * (lxi_m + 1.0) / (lxi_m*lxi_m);
						val3x += mnx;
						val3y += mny;
					}
					
				}//for m
			}
			else if (d <= 0.1 * lenj)
			{
				val0 = -pi * rdi;
				
				if (fabs(s) > 0.5 * lenj)
				{
					mnog1 = 2.0 * rdi * (exp(-fabs(s)  * iDDomRad) * sinh(lenj * iDDomRad / 2.0));
					val3x = mnog1 * normx;
					val3y = mnog1 * normy;
				}
				else
				{
					mnog1 = 2.0 * rdi * (1.0 - exp(-lenj * iDDomRad / 2.0)*cosh(fabs(s) * iDDomRad));
					val3x = mnog1 * normx;
					val3y = mnog1 * normy;			
				}
				break;

			}
		}//if d<50 len 

	}//for j

	i0[locI] = val0;
	i3[2 * locI + 0] = val3x;
	i3[2 * locI + 1] = val3y;
}







__global__ void CU_calc_RHS(
	size_t disp, size_t len, 
	size_t npt, double* pt,
	size_t nvt, double* vt,
	size_t nsr, double* sr,
	double* rhs)
{
	__shared__ double shx[CUBLOCK];
	__shared__ double shy[CUBLOCK];
	__shared__ double shg[CUBLOCK];
	
	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i  = disp + locI;

	int ptindexnext = ((i + 1) < npt) ? i + 1 : 0;


	double begx = pt[i*2 + 0];
	double begy = pt[i*2 + 1];
	double endx = pt[ptindexnext *2 + 0];
	double endy = pt[ptindexnext *2 + 1];

	double dlen = sqrt((endx-begx)*(endx-begx) + (endy-begy)*(endy-begy));

	double val = 0.0;

	double sx, sy, px, py;
	double alpha, lambda, tempVel; //из двух к-тов alpha и lambda в принципе можно для экономии пользоваться одной и той же переменной

	//vortices
	for (size_t j = 0; j < nvt; j += CUBLOCK)
	{
		shx[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = vt[(j + threadIdx.x)*sizeVort + posG + 0];
		
		__syncthreads();

		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			sx = shx[q] - begx;
			sy = shy[q] - begy;

			px = shx[q] - endx;
			py = shy[q] - endy;

			alpha = atan2(px*sy - py*sx, px*sx + py*sy);
		
			tempVel = shg[q] * alpha;
			val -= tempVel;
		}
		__syncthreads();
	}

	//sources	
	for (size_t j = 0; j < nsr; j += CUBLOCK)
	{
		shx[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 0];
		shy[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posR + 1];
		shg[threadIdx.x] = sr[(j + threadIdx.x)*sizeVort + posG + 0] * accelCoeff;

		__syncthreads();
			   
		for (size_t q = 0; q < CUBLOCK; ++q)
		{
			if (j + q < nsr)
			{
				sx = shx[q] - begx;
				sy = shy[q] - begy;

				px = shx[q] - endx;
				py = shy[q] - endy;

				lambda = 0.5*log((sx*sx + sy*sy) / (px*px + py*py));
				
				tempVel = shg[q] * lambda;
				val -= tempVel;
			}
		}
		__syncthreads();
	}

	val *= invdpi / dlen;

	rhs[locI] = val;
}


__global__ void CU_calc_mesh(
	size_t npt, double* pt,
	int* dev_ptr_mesh,
	double meshStep)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	double ptx = pt[i*sizeVort + posR + 0];
	double pty = pt[i*sizeVort + posR + 1];

	dev_ptr_mesh[2 * i + 0] = floor(ptx / meshStep);
	dev_ptr_mesh[2 * i + 1] = floor(pty / meshStep);
}


__global__ void CU_calc_nei(
	size_t disp, size_t len, size_t npt, double* pt,
	int* dev_ptr_mesh, int* dev_ptr_nei,
	double epsCol2, int type)
{
	size_t locI = blockIdx.x * blockDim.x + threadIdx.x;
	size_t i = disp + locI;
	size_t minI = disp + blockIdx.x * blockDim.x;

	if (locI < len)
	{
		int ix = dev_ptr_mesh[2 * i + 0];
		int iy = dev_ptr_mesh[2 * i + 1];

		int jx, jy;

		double ipx, ipy, jpx, jpy;
		ipx = pt[i*sizeVort + posR + 0];
		ipy = pt[i*sizeVort + posR + 1];

		double dx, dy, r2, r2test;
		dev_ptr_nei[locI] = 0;

		double ig = pt[i*sizeVort + posG + 0];
		double jg;

		double cftmax2 = fmax(1.0, (ipx-collapseRightBorder) / collapseScale);
		
		cftmax2 *= cftmax2;

		r2test = (type == 1) ? 4.0*epsCol2 * cftmax2 : epsCol2 * cftmax2;

		//r2test = epsCol2;

		bool cond;

		for (size_t j = minI; j < npt; ++j)
		{
			jx = dev_ptr_mesh[2 * j + 0];
			jy = dev_ptr_mesh[2 * j + 1];

			if ((abs(ix - jx) <= 1) && (abs(iy - jy) <= 1) && (j > i))
			{
				jpx = pt[j*sizeVort + posR + 0];
				jpy = pt[j*sizeVort + posR + 1];

				dx = ipx - jpx;
				dy = ipy - jpy;

				r2 = dx*dx + dy*dy;

				jg = pt[j*sizeVort + posG + 0];

				//printf("max+gamma=%f\n", maxGamma);

				cond = (r2 < r2test) && ((type == 1) ? ig*jg < 0 : (ig*jg > 0) && (fabs(ig + jg) < maxGamma));
				//cond = (r2 < r2test);
				if (cond)
				{
					dev_ptr_nei[locI] = j;
					break;
				}
			}
		}
	}
}

void cuDevice(int n)
{
	hipSetDevice(n);
}


//ниже - обычные (__host__) функции

int cuCalcBlocks(size_t new_n)
{
	size_t nBlocks = new_n / CUBLOCK;
	if (new_n%CUBLOCK)
		nBlocks++;
	return max((int)nBlocks, 1);
}

void cuSetConstants(size_t pos_, size_t posR_, size_t posG_)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(sizeVort), &pos_,  sizeof(size_t));
	hipError_t err2 = hipMemcpyToSymbol(HIP_SYMBOL(posR),     &posR_, sizeof(size_t));
	hipError_t err3 = hipMemcpyToSymbol(HIP_SYMBOL(posG),     &posG_, sizeof(size_t));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetConst01)" << std::endl;
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erSetConst02)" << std::endl;
	if (err3 != hipSuccess)
		std::cout << hipGetErrorString(err3) << " (erSetConst03)" << std::endl;
}

void cuSetAccelCoeff(double cft_)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(accelCoeff), &cft_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetAccelCoeff01)" << std::endl;
}


void cuSetCollapseCoeff(double pos_, double refLength_)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(collapseRightBorder), &pos_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetCollapseCoeff01)" << std::endl;

	hipError_t err2 = hipMemcpyToSymbol(HIP_SYMBOL(collapseScale), &refLength_, sizeof(double));

	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetCollapseCoeff02)" << std::endl;
}


void cuSetMaxGamma(double gam_)
{
	hipError_t err1 = hipMemcpyToSymbol(HIP_SYMBOL(maxGamma), &gam_, sizeof(double));

	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erSetMaxGamma01)" << std::endl;	
}

void cuReserveDevMem(void*& ptr, size_t nBytes)
{
	hipError_t err1 = hipMalloc(&ptr, nBytes);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erReserveDevMem01)" << std::endl;

}

void cuClearWakeMem(size_t new_n, double* dev_ptr)
{
	dim3 blocks(cuCalcBlocks(new_n)), threads(CUBLOCK);
	CU_WakeToZero << <blocks, threads >> > (new_n, dev_ptr);
}

void cuCopyWakeToDev(size_t n, const Vortex2D* host_src, double* dev_ptr)
{
	size_t sizeOfVortex = sizeof(Vortex2D);
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, sizeOfVortex * n, hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyWakeToDev01)" << std::endl;

}

void cuCopyRsToDev(size_t n, const Point2D* host_src, double* dev_ptr)
{
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, sizeof(double)* 2 * n, hipMemcpyHostToDevice);
		
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyRsToDev01)" << std::endl;
}

void cuCopyFixedArray(void* dev_ptr, void* host_src, size_t nBytes)
{
	hipError_t err1 = hipMemcpy(dev_ptr, host_src, nBytes, hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyFixedArray01)" << std::endl;
}

void cuCopyMemFromDev(void* host_ptr, void* dev_ptr, size_t nBytes)
{
	hipError_t err1 = hipMemcpy(host_ptr, dev_ptr, nBytes, hipMemcpyDeviceToHost);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCopyMemFromDev01)" << std::endl;
}

void cuDeleteFromDev(void* devPtr)
{
	hipError_t err1 = hipFree(devPtr);
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erDeleteFromDev01)" << std::endl;
}

/////////////////////////////////////////////////////////////
void cuCalculateConvVeloWake(size_t myDisp, size_t myLen, double* pt, size_t nvt, double* vt, size_t nsr, double* sr, size_t nAfls, size_t* nVtxs, double** ptrVtxs, double* vel, double* rd, double minRd, double eps2, bool onlyRadius)
{	
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);

	if (!onlyRadius)
		CU_calc_conv_epsast << < blocks, threads >> > (myDisp, myLen, pt, nvt, vt, nsr, sr, eps2, minRd, vel, rd, nAfls, nVtxs, ptrVtxs);
	else
		CU_calc_only_epsast << < blocks, threads >> > (myDisp, myLen, pt, nvt, vt, nsr, sr, eps2, minRd, rd, nAfls, nVtxs, ptrVtxs);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_conv_epsast01)" << std::endl;
}


void cuCalculateConvVeloWakeFromVirtual(size_t myDisp, size_t myLen, double* pt, size_t npnl, double* r, double* freegamma, double* attgamma, double* attsource, double* vel, double eps2)
{
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_conv_From_Panels << < blocks, threads >> > (myDisp, myLen, pt, npnl, r, freegamma, attgamma, attsource, eps2, vel);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_conv01)" << std::endl;
}


void cuCalculateDiffVeloWake(size_t myDisp, size_t myLen, double* pt, size_t nvt, double* vt, double* i1, double* i2, double* rd)
{
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_I1I2 << < blocks, threads >> > (myDisp, myLen, pt, nvt, vt, i1, i2, rd);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I1I201)" << std::endl;
}

void cuCalculateDiffVeloWakeMesh(size_t myDisp, size_t myLen, double* pt, size_t nvt, double* vt, int* mesh, double meshStep, double* i1, double* i2, double* rd)
{
	dim3 blocks1(cuCalcBlocks(nvt)), blocks2(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_mesh << < blocks1, threads >> > (nvt, vt, mesh, meshStep);
	
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_mesh01)" << std::endl;

	CU_calc_I1I2mesh << < blocks2, threads >> > (myDisp, myLen, pt, nvt, vt, i1, i2, rd, mesh);

	hipError_t err2 = hipGetLastError();
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erCU_calc_I1I2mesh01)" << std::endl;
}

void cuCalculateDiffVeloWakeFromPanels(size_t myDisp, size_t myLen, double* pt, size_t npnl, double* r, double* freegamma, double* i1, double* i2, double* rd)
{
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_I1I2FromPanels << < blocks, threads >> > (myDisp, myLen, pt, npnl, r, freegamma, i1, i2, rd);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I1I201)" << std::endl;
}


void cuCalculateSurfDiffVeloWake(size_t myDisp, size_t myLen, double* pt, size_t nvt, double* vt, double* i0, double* i3, double* rd)
{
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_I0I3 << < blocks, threads >> > (myDisp, myLen, pt, nvt, vt, i0, i3, rd);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_I0I31)" << std::endl;
}


void cuCalculateRhs(size_t myDisp, size_t myLen, size_t npt, double* pt, size_t nvt, double* vt, size_t nsr, double* sr, double* rhs)
{
	dim3 blocks(cuCalcBlocks(myLen)), threads(CUBLOCK);
	CU_calc_RHS << < blocks, threads >> > (myDisp, myLen, npt, pt, nvt, vt, nsr, sr, rhs);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_RHS01)" << std::endl;
}



void cuCalculatePairs(size_t myDisp, size_t myLen, size_t npt, double* pt, int* mesh, int* nei, double meshStep, double epsCol2, int type)
{
	dim3 blocksMesh(cuCalcBlocks(npt)), threadsMesh(CUBLOCK);
	CU_calc_mesh << < blocksMesh, threadsMesh >> > (npt, pt, mesh, meshStep);

	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess)
		std::cout << hipGetErrorString(err1) << " (erCU_calc_mesh01)" << std::endl;
	
	dim3 blocksNei(cuCalcBlocks(myLen)), threadsNei(CUBLOCK);
	CU_calc_nei << < blocksNei, threadsNei >> > (myDisp, myLen, npt, pt, mesh, nei, epsCol2, type);
	
	hipError_t err2 = hipGetLastError();
	if (err2 != hipSuccess)
		std::cout << hipGetErrorString(err2) << " (erCU_calc_nei01)" << std::endl;
}


/*
void cuTEST(const std::string& str)
{
	double* ppp;
	double mmm = 134;

	hipMalloc(&ppp, 8);

	hipError_t err1 = hipMemcpy(ppp, &mmm, sizeof(double), hipMemcpyHostToDevice);
	if (err1 != hipSuccess)
		std::cout << str << ": " << hipGetErrorString(err1) << " (CUDA_TEST_BREAK)" << std::endl;
	else
		std::cout << str << ": " << hipGetErrorString(err1) << " (CUDA_TEST_PASSED)" << std::endl;
}
*/